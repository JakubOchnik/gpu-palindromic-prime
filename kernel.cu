
#include "hip/hip_runtime.h"


#include <stdio.h>
#include<iostream>
#include<chrono>
#include<fstream>

/*#define imin(a,b) (a<b?a:b)
const int N = 10;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);*/

class countTime {
    bool active;
    std::chrono::steady_clock clk;
    std::chrono::steady_clock::time_point t1;
    std::chrono::steady_clock::time_point t2;
    std::chrono::duration<float> duration;
public:
    countTime() {
        active = false;
    }
    void toggleCount() {
        if (!active) {
            t1 = clk.now();
            active = true;
            return;
        }
        else if (active) {
            t2 = clk.now();
            duration = t2 - t1;
            active = false;
        }
    }
    std::chrono::duration<float> printTime() {
        return duration;
    }
};

__device__ bool isPalindrome(int num) {
    int current = 0, rev = 0;
    int copy = num;
    if (num >= 0 && num <= 9) {
        return true;
    }
    else {
        do {
            current = copy % 10;
            rev = (rev * 10) + current;
            copy = copy / 10;
        } while (copy != 0);
    }
    if (num == rev) {
        return true;
    }
    else
        return false;
}

__global__ void kernel(int* out, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        int i = 1;
        while ((2 * tid + 3) * i + tid + 1 < N) {
            out[(2 * tid + 3) * i + tid + 1] = false;
            i++;
        }
        //__syncthreads();
        if (out[tid] != false && isPalindrome(out[tid])) {
        }
        else
            out[tid] = false;
        tid += gridDim.x * blockDim.x;
    }
}

bool isPalindromeCPU(int num) {
    int current = 0, rev = 0;
    int copy = num;
    if (num >= 0 && num <= 9) {
        return true;
    }
    else {
        do {
            current = copy % 10;
            rev = (rev * 10) + current;
            copy = copy / 10;
        } while (copy != 0);
    }
    if (num == rev) {
        return true;
    }
    else
        return false;
}

void CPUprime(int* out, int N) {
    for (int i = 0; i < N;i++) {
        for (int j = 2; j * j <= i; j++) {
            if (i % j == 0) {
                out[i] = false;
            }
            else if (j + 1 > sqrt(i)) {
                //git
                if (!isPalindromeCPU(out[i])) {
                    out[i] = false;
                }
            }
        }
    }
}

void CPUprimeSieve(bool* out, int N) {
    int i = 2;
    for (; i*i < 2*N; i++) {
        if (out[i] == true) {
            for (int j = i*i; j < 2*N; j += i) {
                out[j] = false;
            }
            //update all next numbers
            if (!isPalindromeCPU(i))
                out[i] = false;
        }
    }
    for (; i < 2 * N; i++) {
        if (out[i] == false || !isPalindromeCPU(i)) {
            out[i] = false;
        }
    }
}

void printSieveArr(bool* arr, int N) {
    for (int i = 0; i < 2*N; i++) {
        if (arr[i] == true)
            printf("%d\n", i);
    }
}

void filterArray(int* arr, int N) {
    for (int i = 0; i < N; i++)
        arr[i] = 2 * i + 1;
    arr[0] = 2;
}

void printArr(int arr[], int N) {
    for (int i = 0; i < N; i++) {
        if (arr[i] != 0)
            printf("%d\n", arr[i]);
    }
}

int main()
{
    std::ofstream data("data.csv");
    if (!data.is_open())
        return -1;
    data.setf(std::ios_base::fixed);
    data << "N,GPU_SIEVE,CPU_SIEVE,CPU_CLASSIC\n";
    for (int i = 10; i <= 100000; i+=10) {
        int N = i;
        printf("%d\n", N);
        data << N << ",";
        const int threadsPerBlock = 256;
        int blocksPerGrid = 32;
        if (32 < (N + threadsPerBlock - 1) / threadsPerBlock)
            int blocksPerGrid = 32;
        else
            int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        int* mainArr = (int*)malloc(N * sizeof(int));
        int* dev_out;

        countTime timer;
        filterArray(mainArr, N);
        hipMalloc((void**)&dev_out, N * sizeof(int));
        hipMemcpy(dev_out, mainArr, N * sizeof(int), hipMemcpyHostToDevice);
        timer.toggleCount();
        kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_out, N);
        timer.toggleCount();
        //printf("A. GPU Sieve of Eratosthenes implementation:\n%f s\n", timer.printTime().count());
        data << timer.printTime().count() << ",";
        hipMemcpy(mainArr, dev_out, N * sizeof(int), hipMemcpyDeviceToHost);
        hipFree(dev_out);

        bool* arr = (bool*)malloc(2 * N * sizeof(bool));
        memset(arr, true, sizeof(arr));

        timer.toggleCount();
        CPUprimeSieve(arr, N);
        timer.toggleCount();
        //printf("B. CPU Sieve of Eratosthenes implementation:\n%f s\n", timer.printTime().count() / 2);
        data << timer.printTime().count()/2 << ",";

        filterArray(mainArr, N);
        timer.toggleCount();
        CPUprime(mainArr, N);
        timer.toggleCount();
        //printf("C. CPU classic:\n%f s\n", timer.printTime().count());
        data << timer.printTime().count() << "\n";

        free(arr);
        free(mainArr);
    }
    data.close();
    return 0;
}
