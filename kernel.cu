
#include "hip/hip_runtime.h"


#include <stdio.h>

#if __INTELLISENSE__
void __syncthreads();
#endif

#define imin(a,b) (a<b?a:b)
const int N = 10;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

int main()
{

    return 0;
}
