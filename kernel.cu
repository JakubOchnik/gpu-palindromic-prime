#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include"cuda_palindromes.h"

#define imin(a,b) (a<b?a:b)
const int threadsPerBlock = 256;

__device__ bool isPalindrome(int num) {
	int current = 0, rev = 0;
	int copy = num;
	if (num >= 0 && num <= 9) {
		return true;
	}
	else {
		do {
			current = copy % 10;
			rev = (rev * 10) + current;
			copy = copy / 10;
		} while (copy != 0);
	}
	if (num == rev)
		return true;
	else
		return false;
}

__global__ void kernel(int* out, int N) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N) {
		int i = 1;
		while ((2 * tid + 3) * i + tid + 1 < N) {
			out[(2 * tid + 3) * i + tid + 1] = false;
			i++;
		}
		if (out[tid] == false || !isPalindrome(out[tid]))
			out[tid] = false;
		tid += gridDim.x * blockDim.x;
	}
}

int* CUDA_palindromes(int* mainArr, int N)
{
	int* dev_out;

	const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

	hipMalloc((void**)&dev_out, N * sizeof(int));
	hipMemcpy(dev_out, mainArr, N * sizeof(int), hipMemcpyHostToDevice);
	kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_out, N);
	hipMemcpy(mainArr, dev_out, N * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_out);
	return 0;
}
